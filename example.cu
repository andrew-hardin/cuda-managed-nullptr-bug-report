
#include <hip/hip_runtime.h>
#include <iostream>

__device__ float CallbackFunction() {
  return 0.0;
}

__managed__ float(*callback_addr)() = &CallbackFunction;

int main() {
  if(callback_addr == nullptr) {
    printf("FAILURE! callback addr = %p\n", callback_addr);
    return EXIT_FAILURE;
  } else {
    printf("Success; callback addr = %p\n", callback_addr);
    return EXIT_SUCCESS;
  }
}